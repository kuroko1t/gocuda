
#include <hip/hip_runtime.h>
extern "C"
{
    __global__ void add(int *c, const int *a, const int *b)
    {
        int i = threadIdx.x;
        c[i] = a[i] + b[i];
    }
}